
#include <hip/hip_runtime.h>
// Device functions
__device__ __host__ float3 distanceBetweenPoints(float3 a, float3 b)
{
	// calculate distance between points
	float3 distance;
	distance.x = b.x - a.x;
	distance.y = b.y - a.y;
	distance.z = b.z - a.z;
	return distance;
}

__device__ __host__ float vectorMagnitude(float3 v)
{
	// calculate magnitude of vector
	return sqrt(v.x * v.x + v.y*v.y + v.z*v.z);
}

__device__ __host__ float3 GetNormal(float4 a, float4 b, float4 c)
{
	float3 x;
	x.x= b.x-a.x;
	x.y= b.y-a.y;
	x.z= b.z-a.z;

	float3 y;
	y.x= c.x-a.x;
	y.y= c.y-a.y;
	y.z= c.z-a.z;

	float3 n= make_float3(-(y.y*x.z- y.z*x.y), -(y.z*x.x- y.x*x.z), -(y.x*x.y - y.y*x.x));
	return n;
}
